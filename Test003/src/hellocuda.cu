
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
 
#define N 16
const int blocksize = 4; 
 
__global__ 
void init_random(hiprandState_t *state) 
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(0,idx,0,&state[idx]);
}

__global__ 
void generate_random(hiprandState_t *state, double *b) 
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N){
		b[idx] = hiprand_normal_double(&state[idx]);
	}
}
 
int main()
{

	hipError_t ierrAsync;
    hipError_t ierrSync;

	double *b;
	hiprandState_t *state;
	
	// allocate memory in GPU for random number creation state objects.
	hipMallocManaged(&state, N * sizeof(hiprandState_t));
	hipMallocManaged(&b, N * sizeof(double));
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( N/blocksize+1, 1 );
	init_random<<<dimGrid, dimBlock>>>(state);
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize();
	if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

	generate_random<<<dimGrid, dimBlock>>>(state,b);
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize();
	if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

	for (int i=0; i<N; i++){
		std::cout << b[i] << std::endl;
	}

	hipFree( state );
	hipFree( b );
	
	return EXIT_SUCCESS;
}
